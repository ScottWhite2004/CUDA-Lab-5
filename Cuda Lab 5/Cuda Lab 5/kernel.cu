﻿
#include "hip/hip_runtime.h"


#include <stdio.h>


int main()
{
    const int heightA = 4;
    const int widthA = 3;
    const int heightB = 3;
    const int widthB = 2;

    const int matrixA[heightA][widthA] = {
        {1,2,3},
        {1,2,3},
        {1,2,3},
        {1,2,3}
    };
    const int matrixB[heightB][widthB] = {
        {1,2},
        {1,2},
        {1,2}
    };
    const int arraySizeC = heightA * widthB;
    int matrixC[heightA][widthB];

    for (int i = 0; i < heightA; i++)
    {
        for (int j = 0; j < widthB; j++)
        {
            matrixC[i][j] = 0;
            for (int idx = 0; idx < widthA; idx++)
            {
                matrixC[i][j] += matrixA[i][idx] * matrixB[idx][j];
            }
        }
    }

    printf("Matrix %d %d %d %d %d %d %d %d", matrixC[0][0], matrixC[0][1], matrixC[1][0], matrixC[1][1], matrixC[2][0], matrixC[2][1], matrixC[3][0], matrixC[3][1]);
    return 0;
}


