#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void addKernel(int *c, const int *a, const int *b, const int *widthA,const int *widthB)
{
    int i = threadIdx.y;
    int j = threadIdx.x;
    int C_ij = i * blockDim.x + j;

    int tempC = 0;
    for (int k = 0; k < *widthA; k++)
    {
        int i_A = i * *widthA + k;
        int i_B = j + k * *widthB;
        tempC += a[i_A] * b[i_B];
    }
    c[C_ij] = tempC;
    
}

int main()
{
    
    const int heightA = 4;
    const int widthA = 3;
    const int heightB = 3;
    const int widthB = 2;
    const int arraySizeA = heightA * widthA;
    const int arraySizeB = heightB * widthB;
    const int arraySizeC = heightA * widthB;

    const int a[arraySizeA] = { 1,2,3,1,2,3,1,2,3,1,2,3 };
    const int b[arraySizeB] = { 1,2,1,2,1,2 };
    int c[arraySizeC] = { 0 };
    
    int* dev_a, * dev_b, * dev_c;
    int* dev_widA, *dev_widB;

    hipMalloc(&dev_a, arraySizeA * sizeof(int));
    hipMalloc(&dev_b, arraySizeB * sizeof(int));
    hipMalloc(&dev_c, arraySizeC * sizeof(int));
    hipMalloc(&dev_widA, sizeof(int));
    hipMalloc(&dev_widB, sizeof(int));

    hipMemcpy(dev_a, a, arraySizeA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySizeB * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_widA, &widthA, arraySizeA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_widB, &widthB, arraySizeB * sizeof(int), hipMemcpyHostToDevice);

    addKernel << <8,8>> > (dev_a, dev_b, dev_c, dev_widA, dev_widB);

    hipMemcpy(c, dev_c, arraySizeC * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    printf("Dot product %d", c[1]);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipFree(dev_widA);
    hipFree(dev_widB);
    
    return 0;
}


