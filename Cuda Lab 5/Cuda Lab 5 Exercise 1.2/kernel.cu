#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void addKernel(int *c, int *a, int *b, int widthA,int widthB)
{
    int i = threadIdx.y;
    int j = threadIdx.x;
    int C_ij = i * blockDim.x + j;

    int tempC = 0;
    for (int k = 0; k < widthA; k++)
    {
        int i_A = i * widthA + k;
        int i_B = j + k * widthB;
        tempC += a[i_A] * b[i_B];
    }
    c[C_ij] = tempC;
}

int main()
{
    
    const int heightA = 4;
    int widthA = 3;
    const int heightB = 3;
    int widthB = 2;
    int heightD = 6;
    int widthD = 4;
    int heightE = 4;
    int widthE = 3;
    const int arraySizeD = heightD * widthD;
    const int arraySizeE = heightE * widthE;
    const int arraySizeF = heightD * widthE;
    const int arraySizeA = heightA * widthA;
    const int arraySizeB = heightB * widthB;
    const int arraySizeC = heightA * widthB;

    int* a, * b, * c, * d, * e, * f;

    hipMallocManaged(&a, arraySizeA * sizeof(int));
    hipMallocManaged(&b, arraySizeB * sizeof(int));
    hipMallocManaged(&c, arraySizeC * sizeof(int));
    hipMallocManaged(&d, arraySizeD * sizeof(int));
    hipMallocManaged(&e, arraySizeE * sizeof(int));
    hipMallocManaged(&f, arraySizeF * sizeof(int));

    for (int i = 0; i < arraySizeA; i++)
    {
        a[i] = i;
    }

    for (int i = 0; i < arraySizeB; i++)
    {
        b[i] = i;
    }
    for (int i = 0; i < arraySizeD; i++)
    {
        d[i] = i;
    }
    for (int i = 0; i < arraySizeE; i++)
    {
        e[i] = i;
    }

    addKernel << <1,dim3(2,4) >> > (c, a, b, widthA, widthB);
    addKernel << <1, dim3(3, 6) >> > (f, d, e, widthD, widthE);

    hipDeviceSynchronize();



    printf("Dot product Matrix a and b \n%d %d \n%d %d \n%d %d \n%d %d\n", c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7]);
    printf("Dot product Matrix d and e \n%d %d %d \n%d %d %d \n%d %d %d \n%d %d %d \n%d %d %d \n%d %d %d\n ", f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14], f[15], f[16], f[17]);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    hipFree(d);
    hipFree(e);
    hipFree(f);


    
    return 0;
}


