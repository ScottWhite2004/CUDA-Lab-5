#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void addKernel(int *c, int *a, int *b, int widthA,int widthB)
{
    int i = threadIdx.y;
    int j = threadIdx.x;
    int C_ij = i * blockDim.x + j;

    int tempC = 0;
    for (int k = 0; k < widthA; k++)
    {
        int i_A = i * widthA + k;
        int i_B = j + k * widthB;
        tempC += a[i_A] * b[i_B];
    }
    c[C_ij] = tempC;
}

int main()
{
    
    const int heightA = 4;
    int widthA = 3;
    const int heightB = 3;
    int widthB = 2;
    const int arraySizeA = heightA * widthA;
    const int arraySizeB = heightB * widthB;
    const int arraySizeC = heightA * widthB;

    int* a, * b, * c;

    hipMallocManaged(&a, arraySizeA * sizeof(int));
    hipMallocManaged(&b, arraySizeB * sizeof(int));
    hipMallocManaged(&c, arraySizeC * sizeof(int));

    for (int i = 0; i < arraySizeA; i++)
    {
        a[i] = i;
    }

    for (int i = 0; i < arraySizeB; i++)
    {
        b[i] = i;
    }

    for (int i = 0; i < arraySizeC; i++)
    {
        c[i] = 0;
    }

    addKernel << <1,dim3(2,4) >> > (c, a, b, widthA, widthB);

    hipDeviceSynchronize();



    printf("Dot product %d %d \n %d %d \n %d %d \n %d %d", c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7]);

    hipFree(a);
    hipFree(b);
    hipFree(c);


    
    return 0;
}


